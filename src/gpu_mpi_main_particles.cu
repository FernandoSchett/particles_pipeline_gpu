#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <ctime>
#include <sys/stat.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>
#include <mpi.h>

#include "helper.hpp"

#define DEFAULT_POWER 3

void parse_args(int argc, char **argv, int *power, dist_type_t *dist_type)
{
    *power = DEFAULT_POWER;
    *dist_type = DIST_UNKNOWN;

    if (argc > 1)
    {
        if (std::strcmp(argv[1], "box") == 0)
            *dist_type = DIST_BOX;
        else if (std::strcmp(argv[1], "torus") == 0)
            *dist_type = DIST_TORUS;
    }

    if (*dist_type == DIST_UNKNOWN)
    {
        *dist_type = DIST_BOX;
    }

    if (argc > 2)
    {
        *power = std::atoi(argv[2]);
    }
}

void distribute_gpu_particles_mpi(t_particle **d_rank_array, int *lens, int *capacity, hipStream_t stream);

int main(int argc, char **argv)
{
    MPI_Init(&argc, &argv);

    int rank = 0;
    int nprocs = 1;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);

    int gpus = 0;
    hipGetDeviceCount(&gpus);
    int local_dev = rank % (gpus > 0 ? gpus : 1);
    hipSetDevice(local_dev);

    if (rank == 0)
        std::cout << "Using " << nprocs << " GPUs\n";
    char filename[128];

    int length_per_rank = 0;
    long long total_particles = 0;

    dist_type_t dist_type;
    int power = DEFAULT_POWER;
    double box_length = 0.0;
    int major_r = 0;
    int minor_r = 0;
    double RAM_GB = 0.0;
    int capacity = 0;

    const int block = 256;
    int sms = 0;

    parse_args(argc, argv, &power, &dist_type);

    t_particle *d_rank_array = nullptr;
    t_particle *h_host_array = nullptr;
    hipStream_t gpu_stream;
    int lens = 0;

    setup_particles_box_length(power, nprocs, rank, &length_per_rank, &box_length, &total_particles, &RAM_GB, &major_r, &minor_r);
    lens = length_per_rank;
    printf("Before distribution %d:  %d\n", rank, lens);

    hipStreamCreate(&gpu_stream);

    hipMallocAsync(&d_rank_array, length_per_rank * sizeof(t_particle), gpu_stream);
    hipHostMalloc(&h_host_array, length_per_rank * sizeof(t_particle));
    capacity = length_per_rank;

    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, local_dev);
    int maxBlocks = sms * 20;
    int grid = (length_per_rank + block - 1) / block;
    int seed = rank;
    if (grid > maxBlocks)
        grid = maxBlocks;

    switch (dist_type)
    {
    case DIST_BOX:
        box_distribution_kernel<<<grid, block, 0, gpu_stream>>>(d_rank_array, length_per_rank, box_length, seed);
        break;
    case DIST_TORUS:
        torus_distribution_kernel<<<grid, block, 0, gpu_stream>>>(d_rank_array, length_per_rank, major_r, minor_r, box_length, seed);
        break;
    }

    hipStreamSynchronize(gpu_stream);
    MPI_Barrier(MPI_COMM_WORLD);

    auto t0 = std::chrono::steady_clock::now();

    generate_keys_kernel<<<grid, block, 0, gpu_stream>>>(d_rank_array, length_per_rank, box_length);

    hipStreamSynchronize(gpu_stream);
    MPI_Barrier(MPI_COMM_WORLD);

    distribute_gpu_particles_mpi(&d_rank_array, &length_per_rank, &capacity, gpu_stream);

    hipStreamSynchronize(gpu_stream);
    MPI_Barrier(MPI_COMM_WORLD);

    auto t1 = std::chrono::steady_clock::now();
    double dist_sec = std::chrono::duration<double>(t1 - t0).count();

    lens = length_per_rank;

    if (power < 4)
    {
        if (h_host_array)
        {
            hipHostFree(h_host_array);
            h_host_array = nullptr;
        }

        const size_t bytes = static_cast<size_t>(lens) * sizeof(t_particle);
        if (lens > 0)
        {
            hipHostMalloc(&h_host_array, bytes);
            hipMemcpyAsync(h_host_array, d_rank_array, bytes, hipMemcpyDeviceToHost, gpu_stream);
        }

        hipStreamSynchronize(gpu_stream);
        MPI_Barrier(MPI_COMM_WORLD);

        std::vector<int> recv_lens;
        if (rank == 0)
            recv_lens.resize(nprocs);
        MPI_Gather(&lens, 1, MPI_INT, rank == 0 ? recv_lens.data() : nullptr, 1, MPI_INT, 0, MPI_COMM_WORLD);

        std::vector<int> recv_counts, recv_displs;
        size_t total_count = 0;
        if (rank == 0)
        {
            recv_counts.resize(nprocs);
            recv_displs.resize(nprocs);
            for (int i = 0; i < nprocs; ++i)
            {
                recv_counts[i] = recv_lens[i] * (int)sizeof(t_particle);
            }
            recv_displs[0] = 0;
            for (int i = 1; i < nprocs; ++i)
                recv_displs[i] = recv_displs[i - 1] + recv_counts[i - 1];
            total_count = (size_t)recv_displs.back() + (size_t)recv_counts.back();
        }

        std::vector<unsigned char> gather_buf(rank == 0 ? total_count : 0);
        MPI_Gatherv(d_rank_array, lens * (int)sizeof(t_particle), MPI_BYTE,
                    rank == 0 ? gather_buf.data() : nullptr,
                    rank == 0 ? recv_counts.data() : nullptr,
                    rank == 0 ? recv_displs.data() : nullptr,
                    MPI_BYTE, 0, MPI_COMM_WORLD);

        if (rank == 0)
        {
            sprintf(filename, "particle_file_gpu_n%d_total%lld.par", nprocs, total_particles);
            std::vector<t_particle *> host_ptrs(nprocs, nullptr);
            for (int i = 0; i < nprocs; ++i)
                host_ptrs[i] = reinterpret_cast<t_particle *>(gather_buf.data() + recv_displs[i]);
            int rc = concat_and_serial_write(host_ptrs.data(), recv_lens.data(), nprocs, filename);
            if (rc != 0)
            {
                std::cerr << "Error at writing file, rc=" << rc << "\n";
            }
        }
    }

    log_results(rank, power, total_particles, length_per_rank, nprocs, box_length, RAM_GB, dist_sec, "gpu");

    if (d_rank_array)
        hipFreeAsync(d_rank_array, gpu_stream);
    if (h_host_array)
        hipHostFree(h_host_array);
    hipStreamDestroy(gpu_stream);

    MPI_Barrier(MPI_COMM_WORLD);
    MPI_Finalize();
    return 0;
}
