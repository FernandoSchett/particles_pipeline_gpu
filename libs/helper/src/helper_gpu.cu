#include "hip/hip_runtime.h"
#include "./helper.hpp"

__global__ void box_distribution_kernel(t_particle *particles, int N, double L, unsigned long long seed)
{
    using RNG = r123::Philox4x32;
    RNG::key_type key = {{(uint32_t)seed, (uint32_t)(seed >> 32)}};

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        RNG::ctr_type ctr = {{(uint32_t)i, 0u, 0u, 0u}};
        RNG::ctr_type r = RNG()(ctr, key);

        particles[i].coord[0] = r123::u01<double>(r.v[0]) * L;
        particles[i].coord[1] = r123::u01<double>(r.v[1]) * L;
        particles[i].coord[2] = r123::u01<double>(r.v[2]) * L;
    }
}

__global__ void torus_distribution_kernel(t_particle *particles, int N, double major_r, double minor_r, double box_length, unsigned long long seed)
{
    using RNG = r123::Philox4x32;
    RNG::key_type key = {{(uint32_t)seed, (uint32_t)(seed >> 32)}};
    const double TWO_PI = 6.283185307179586476925286766559;
    const double center = box_length * 0.5;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        RNG::ctr_type ctr = {{(uint32_t)i, 0u, 0u, 0u}};
        RNG::ctr_type rnum = RNG()(ctr, key);

        double u0 = r123::u01<double>(rnum.v[0]);
        double u1 = r123::u01<double>(rnum.v[1]);
        double u2 = r123::u01<double>(rnum.v[2]);

        double theta = TWO_PI * u0;
        double phi = TWO_PI * u1;
        double r = minor_r * sqrt(u2);

        double cphi = cos(phi);
        double sphi = sin(phi);
        double cth = cos(theta);
        double sth = sin(theta);

        double Rplus = major_r + r * cphi;

        particles[i].coord[0] = center + Rplus * cth;
        particles[i].coord[1] = center + Rplus * sth;
        particles[i].coord[2] = center + r * sphi;
    }
}

__global__ void generate_keys_kernel(t_particle *particles, int N, double box_length)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        double x = particles[i].coord[0];
        double y = particles[i].coord[1];
        double z = particles[i].coord[2];

        double ox = 0.0, oy = 0.0, oz = 0.0;
        double len = box_length;

        unsigned long long key = 0ull;

#pragma unroll 1
        for (int d = 0; d < MAX_DEPTH; ++d)
        {
            len *= 0.5;
            int oct = 0;

            double cx = ox + len;
            double cy = oy + len;
            double cz = oz + len;

            if (x >= cx)
            {
                oct |= 1;
                ox += len;
            }
            if (y >= cy)
            {
                oct |= 2;
                oy += len;
            }
            if (z >= cz)
            {
                oct |= 4;
                oz += len;
            }

            key = (key << 3) | (unsigned long long)oct;
        }

        particles[i].key = (long long)key;
    }
}

__global__ void set_rank_kernel(t_particle *p, int n, int rank_id)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        p[i].mpi_rank = rank_id;
}

void gpu_barrier(int nprocs, const std::vector<hipStream_t> &streams)
{
    for (int d = 0; d < nprocs; ++d)
    {
        hipSetDevice(d);
        hipStreamSynchronize(streams[d]);
    }
}

void enable_p2p_all(int ndev)
{
    for (int i = 0; i < ndev; ++i)
    {
        hipSetDevice(i);
        for (int j = 0; j < ndev; ++j)
        {
            if (i == j)
                continue;

            int can = 0;
            hipDeviceCanAccessPeer(&can, i, j);

            if (can)
            {
                auto err = hipDeviceEnablePeerAccess(j, 0);
                if (err != hipSuccess && err != hipErrorPeerAccessAlreadyEnabled)
                {
                    hipGetLastError();
                }
            }
        }
    }
}

int concat_and_serial_write(t_particle **arrays, const int *counts, int nprocs, const char *filename)
{
    long long total_ll = 0;
    for (int d = 0; d < nprocs; ++d)
    {
        if (counts[d] < 0)
            return 1;
        total_ll += (long long)counts[d];
    }

    if (total_ll > std::numeric_limits<int>::max())
    {
        std::fprintf(stderr, "[E] total particles > INT_MAX (%lld)\n", total_ll);
        return 2;
    }
    const int total = (int)total_ll;

    std::vector<t_particle> tmp;
    tmp.reserve((size_t)total);

    for (int d = 0; d < nprocs; ++d)
    {
        const int n = counts[d];
        if (n <= 0)
            continue;
        if (!arrays[d])
            return 3;

        tmp.insert(tmp.end(), arrays[d], arrays[d] + n);
    }
    return serial_write_to_file(tmp.data(), total, const_cast<char *>(filename));
}

inline void compute_cuts_for_dev(
    int dev,
    t_particle *d_ptr,
    int n,
    const std::vector<unsigned long long> &splitters,
    std::vector<int> &cuts,
    hipStream_t stream)
{
    hipSetDevice(dev);

    cuts.assign(splitters.size() + 2, 0);
    cuts[0] = 0;

    if (n <= 0)
    {
        cuts.back() = 0;
        return;
    }

    thrust::device_ptr<t_particle> first(d_ptr), last(d_ptr + n);
    auto pol = thrust::cuda::par.on(stream);

    for (size_t b = 0; b < splitters.size(); ++b)
    {
        t_particle probe;
        probe.key = (long long)splitters[b];
        auto it = thrust::upper_bound(pol, first, last, probe, key_less{});
        cuts[b + 1] = static_cast<int>(it - first);
    }

    cuts.back() = n;
}

long long count_leq_device(
    int dev, t_particle *d_ptr, int n, unsigned long long mid, hipStream_t stream)
{
    if (n <= 0)
        return 0;
    hipSetDevice(dev);
    t_particle probe;
    probe.key = (long long)mid;
    auto pol = thrust::cuda::par.on(stream);
    thrust::device_ptr<t_particle> first(d_ptr), last(d_ptr + n);
    auto it = thrust::upper_bound(pol, first, last, probe, key_less{});
    return static_cast<long long>(it - first);
}

int distribute_gpu_particles(std::vector<t_particle *> &d_rank_array, std::vector<int> &lens, std::vector<hipStream_t> &gpu_streams)
{
    const int nprocs = (int)d_rank_array.size();
    for (int dev = 0; dev < nprocs; ++dev)
    {
        hipSetDevice(dev);
        int n = lens[dev];
        if (n <= 0)
            continue;
        auto pol = thrust::cuda::par.on(gpu_streams[dev]);
        thrust::device_ptr<t_particle> first(d_rank_array[dev]), last(d_rank_array[dev] + n);
        thrust::sort(pol, first, last, key_less{});
    }

    gpu_barrier(nprocs, gpu_streams);

    std::vector<unsigned long long> local_min(nprocs, std::numeric_limits<unsigned long long>::max());
    std::vector<unsigned long long> local_max(nprocs, 0ull);
    for (int dev = 0; dev < nprocs; ++dev)
    {
        int n = lens[dev];
        if (n <= 0)
            continue;
        t_particle first_h{}, last_h{};
        hipMemcpy(&first_h, d_rank_array[dev], sizeof(t_particle), hipMemcpyDeviceToHost);
        hipMemcpy(&last_h, d_rank_array[dev] + (n - 1), sizeof(t_particle), hipMemcpyDeviceToHost);
        local_min[dev] = (unsigned long long)first_h.key;
        local_max[dev] = (unsigned long long)last_h.key;
    }

    unsigned long long gmin = std::numeric_limits<unsigned long long>::max();
    unsigned long long gmax = 0ull;
    long long N_global = 0;
    for (int dev = 0; dev < nprocs; ++dev)
    {
        if (lens[dev] > 0)
        {
            gmin = std::min(gmin, local_min[dev]);
            gmax = std::max(gmax, local_max[dev]);
            N_global += lens[dev];
        }
    }
    if (N_global == 0)
        return 0;

    std::vector<unsigned long long> splitters;
    splitters.reserve(nprocs ? nprocs - 1 : 0);
    unsigned long long lo_base = gmin;
    for (int i = 1; i < nprocs; ++i)
    {
        const long long target = (N_global * i + nprocs - 1) / nprocs;
        unsigned long long lo = lo_base, hi = gmax;
        long long c_global = 0;
        while (lo < hi)
        {
            unsigned long long mid = lo + ((hi - lo) >> 1);
            c_global = 0;
            for (int dev = 0; dev < nprocs; ++dev)
            {
                if (lens[dev] == 0)
                    continue;
                c_global += count_leq_device(dev, d_rank_array[dev], lens[dev], mid, gpu_streams[dev]);
            }
            if (c_global >= target)
                hi = mid;
            else
                lo = mid + 1;
        }
        // printf("C GLOBAL: %lld\n", c_global);
        // printf("TARGET: %lld\n", target);
        // printf("lo: %lu\n", lo);
        // printf("high: %lu\n", hi);

        splitters.push_back(lo);
        lo_base = lo;
    }

    std::vector<std::vector<int>> sendcounts(nprocs, std::vector<int>(nprocs, 0));
    std::vector<std::vector<int>> cuts(nprocs);
    for (int src = 0; src < nprocs; ++src)
    {
        if (lens[src] == 0)
        {
            cuts[src] = std::vector<int>(nprocs + 1, 0);
            continue;
        }
        compute_cuts_for_dev(src, d_rank_array[src], lens[src], splitters, cuts[src], gpu_streams[src]);
    }

    gpu_barrier(nprocs, gpu_streams);

    for (int src = 0; src < nprocs; ++src)
        for (int b = 0; b < nprocs; ++b)
        {
            int begin = cuts[src][b];
            int end = cuts[src][b + 1];
            sendcounts[src][b] = std::max(0, end - begin);
        }

    std::vector<std::vector<int>> recvcounts(nprocs, std::vector<int>(nprocs, 0));
    for (int dst = 0; dst < nprocs; ++dst)
        for (int src = 0; src < nprocs; ++src)
            recvcounts[dst][src] = sendcounts[src][dst];

    auto prefix = [&](const std::vector<int> &v)
    {
        std::vector<int> p(v.size(), 0);
        for (size_t i = 1; i < v.size(); ++i)
            p[i] = p[i - 1] + v[i - 1];
        return p;
    };

    std::vector<std::vector<int>> sdispls(nprocs), rdispls(nprocs);
    for (int src = 0; src < nprocs; ++src)
        sdispls[src] = prefix(sendcounts[src]);
    for (int dst = 0; dst < nprocs; ++dst)
        rdispls[dst] = prefix(recvcounts[dst]);

    std::vector<int> recv_tot(nprocs, 0);
    for (int dst = 0; dst < nprocs; ++dst)
        recv_tot[dst] = std::accumulate(recvcounts[dst].begin(), recvcounts[dst].end(), 0);

    std::vector<t_particle *> d_new(nprocs, nullptr);
    for (int dst = 0; dst < nprocs; ++dst)
    {
        hipSetDevice(dst);
        if (recv_tot[dst] > 0)
            hipMallocAsync(&d_new[dst], (size_t)recv_tot[dst] * sizeof(t_particle), gpu_streams[dst]);
    }

    for (int src = 0; src < nprocs; ++src)
        for (int dst = 0; dst < nprocs; ++dst)
        {
            int cnt = sendcounts[src][dst];
            if (cnt <= 0)
                continue;
            const int begin_src = cuts[src][dst];
            const size_t bytes = (size_t)cnt * sizeof(t_particle);
            t_particle *src_ptr = d_rank_array[src] + begin_src;
            t_particle *dst_ptr = d_new[dst] + rdispls[dst][src];
            hipSetDevice(dst);
            hipMemcpyPeerAsync(dst_ptr, dst, src_ptr, src, bytes, gpu_streams[dst]);
        }

    gpu_barrier(nprocs, gpu_streams);

    for (int dev = 0; dev < nprocs; ++dev)
    {
        hipSetDevice(dev);
        if (d_rank_array[dev])
            hipFreeAsync(d_rank_array[dev], gpu_streams[dev]);
        d_rank_array[dev] = d_new[dev];
        lens[dev] = recv_tot[dev];
    }

    const int block = 256;
    for (int dev = 0; dev < nprocs; ++dev)
    {
        hipSetDevice(dev);
        int n = lens[dev];
        printf("After distribution %d:  %d\n", dev, lens[dev]);
        if (n == 0)
            continue;
        int grid = (n + block - 1) / block;
        set_rank_kernel<<<grid, block, 0, gpu_streams[dev]>>>(d_rank_array[dev], n, dev);
    }

    gpu_barrier(nprocs, gpu_streams);
    return 0;
}

static long long count_leq_device2(t_particle *d_ptr, int n, unsigned long long key, hipStream_t stream)
{
    if (n <= 0)
        return 0;
    t_particle probe;
    probe.key = (long long)key;
    auto pol = thrust::cuda::par.on(stream);
    thrust::device_ptr<t_particle> first(d_ptr), last(d_ptr + n);
    auto it = thrust::upper_bound(pol, first, last, probe, key_less{});
    return static_cast<long long>(it - first);
}

void distribute_gpu_particles_mpi(t_particle **d_rank_array, int *lens, int *capacity, hipStream_t stream)
{
    int rank, nprocs;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);

    {
        auto pol = thrust::cuda::par.on(stream);
        thrust::device_ptr<t_particle> first(*d_rank_array), last(*d_rank_array + *lens);
        thrust::sort(pol, first, last, key_less{});
    }

    unsigned long long local_min = std::numeric_limits<unsigned long long>::max();
    unsigned long long local_max = 0ull;
    if (*lens > 0)
    {
        t_particle a, b;
        hipMemcpyAsync(&a, *d_rank_array, sizeof(t_particle), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(&b, *d_rank_array + (*lens - 1), sizeof(t_particle), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        local_min = (unsigned long long)a.key;
        local_max = (unsigned long long)b.key;
    }

    unsigned long long gmin = 0ull, gmax = 0ull;
    MPI_Allreduce(&local_min, &gmin, 1, MPI_UNSIGNED_LONG_LONG, MPI_MIN, MPI_COMM_WORLD);
    MPI_Allreduce(&local_max, &gmax, 1, MPI_UNSIGNED_LONG_LONG, MPI_MAX, MPI_COMM_WORLD);

    long long N_local = *lens, N_global = 0;
    MPI_Allreduce(&N_local, &N_global, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
    if (N_global == 0)
        return;

    std::vector<unsigned long long> splitters;
    splitters.reserve(nprocs ? nprocs - 1 : 0);
    unsigned long long lo_base = gmin;
    for (int i = 1; i < nprocs; ++i)
    {
        const long long target = (N_global * i + nprocs - 1) / nprocs;
        unsigned long long lo = lo_base, hi = gmax;
        while (lo < hi)
        {
            const unsigned long long mid = lo + ((hi - lo) >> 1);
            long long cnt_local = count_leq_device2(*d_rank_array, *lens, mid, stream);
            long long cnt_global = 0;
            MPI_Allreduce(&cnt_local, &cnt_global, 1, MPI_LONG_LONG_INT, MPI_SUM, MPI_COMM_WORLD);
            if (cnt_global >= target)
                hi = mid;
            else
                lo = mid + 1;
        }
        splitters.push_back(lo);
        lo_base = lo;
    }

    std::vector<long long> pos(nprocs + 1, 0);
    {
        auto pol = thrust::cuda::par.on(stream);
        thrust::device_ptr<t_particle> first(*d_rank_array), last(*d_rank_array + *lens);
        for (int i = 0; i < nprocs - 1; ++i)
        {
            t_particle probe;
            probe.key = (long long)splitters[i];
            auto it = thrust::upper_bound(pol, first, last, probe, key_less{});
            pos[i + 1] = static_cast<long long>(it - first);
        }
        pos[nprocs] = *lens;
    }

    std::vector<int> send_counts(nprocs, 0);
    for (int r = 0; r < nprocs; ++r)
    {
        long long start = pos[r];
        long long end = pos[r + 1];
        long long c = end - start;
        send_counts[r] = (c > 0) ? static_cast<int>(c) : 0;
    }

    std::vector<int> recv_counts(nprocs, 0);
    MPI_Alltoall(send_counts.data(), 1, MPI_INT, recv_counts.data(), 1, MPI_INT, MPI_COMM_WORLD);

    std::vector<int> send_displs(nprocs, 0), recv_displs(nprocs, 0);
    for (int i = 1; i < nprocs; ++i)
    {
        send_displs[i] = send_displs[i - 1] + send_counts[i - 1];
        recv_displs[i] = recv_displs[i - 1] + recv_counts[i - 1];
    }
    const int total_send = send_displs.back() + send_counts.back();
    const int total_recv = recv_displs.back() + recv_counts.back();

    t_particle *d_tmp = nullptr;
    if (total_recv > 0)
        hipMallocAsync(&d_tmp, (size_t)total_recv * sizeof(t_particle), stream);
    else
        hipMallocAsync(&d_tmp, 1, stream);

    std::vector<int> send_counts_bytes(nprocs), recv_counts_bytes(nprocs), send_displs_bytes(nprocs), recv_displs_bytes(nprocs);
    for (int i = 0; i < nprocs; ++i)
    {
        send_counts_bytes[i] = send_counts[i] * (int)sizeof(t_particle);
        recv_counts_bytes[i] = recv_counts[i] * (int)sizeof(t_particle);
        send_displs_bytes[i] = send_displs[i] * (int)sizeof(t_particle);
        recv_displs_bytes[i] = recv_displs[i] * (int)sizeof(t_particle);
    }

    hipStreamSynchronize(stream);
    MPI_Alltoallv(
        *d_rank_array, send_counts_bytes.data(), send_displs_bytes.data(), MPI_BYTE,
        d_tmp, recv_counts_bytes.data(), recv_displs_bytes.data(), MPI_BYTE,
        MPI_COMM_WORLD);

    if (total_recv > *capacity)
    {
        if (*d_rank_array)
            hipFree(*d_rank_array);
        hipMalloc(d_rank_array, (size_t)total_recv * sizeof(t_particle));
        *capacity = total_recv;
    }

    if (total_recv > 0)
    {
        hipMemcpyAsync(*d_rank_array, d_tmp, (size_t)total_recv * sizeof(t_particle), hipMemcpyDeviceToDevice, stream);
    }
    *lens = total_recv;

    hipStreamSynchronize(stream);
    hipFreeAsync(d_tmp, stream);

    if (*lens > 0)
    {
        const int block = 256;
        const int grid = (*lens + block - 1) / block;
        set_rank_kernel<<<grid, block, 0, stream>>>(*d_rank_array, *lens, rank);
        hipStreamSynchronize(stream);
    }

    printf("rank %d: %d\n", rank, *lens);
}
